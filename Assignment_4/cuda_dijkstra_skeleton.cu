#include "hip/hip_runtime.h"
/* Name:
 * ID:
 * Email:
 */

/*
 * This is code skeleton for COMP5112-17Spring assignment4
 * Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_dijkstra cuda_dijkstra_skeleton.cu
 * Run: ./cuda_dijkstra -n <number of threads> -i <input file>,
 * you will find the output in 'output.txt' file
 *
 *  by Lipeng WANG, 5th Apr 2017
 */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <vector>
#include <climits>
#include <cstring>
#include <cmath>
#include <algorithm>
#include <sys/time.h>
#include <time.h>
#include <getopt.h>

#include <hip/hip_runtime.h>
#include <>

using std::string;
using std::cout;
using std::endl;
using std::vector;
using std::ceil;
using std::memcpy;

#define INF 1000000

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
 {                                                                              \
     const hipError_t error = call;                                            \
     if (error != hipSuccess)                                                  \
     {                                                                          \
         fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
         fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                 hipGetErrorString(error));                                    \
         exit(1);                                                               \
     }                                                                          \
 }

/*
 * utils is a namespace for utility functions
 * including I/O (read input file and print results) and one matrix dimension convert(2D->1D) function
 */
namespace utils {
    int num_threads; //number of thread
    int N; //number of vertices
    int *mat; // the adjacency matrix

    string filename; // input file name
    string outputfile; //output file name, default: 'output.txt'

    void print_usage() {
        cout << "Usage:\n" << "\tcuda_dijkstra -n <number of threads per block> -i <input file>" << endl;
        exit(0);
    }

    int parse_args(int argc, char **argv) {
        filename = "";
        outputfile = "output.txt";
        num_threads = 0;

        int opt;
        if (argc < 2) {
            print_usage();
        }
        while ((opt = getopt(argc, argv, "n:i:o:h")) != EOF) {
            switch (opt) {
                case 'n':
                    num_threads = atoi(optarg);
                    break;
                case 'i':
                    filename = optarg;
                    break;
                case 'o':
                    outputfile = optarg;
                    break;
                case 'h':
                case '?':
                default:
                    print_usage();
            }
        }
        if (filename.length() == 0 || num_threads == 0)
            print_usage();
        return 0;
    }

    /*
     * convert 2-dimension coordinate to 1-dimension
     */
    int convert_dimension_2D_1D(int x, int y) {
        return x * N + y;
    }

    int read_file(string filename) {
        std::ifstream inputf(filename, std::ifstream::in);
        inputf >> N;
        assert(N < (1024 * 1024 *
                    20)); // input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
        mat = (int *) malloc(N * N * sizeof(int));
        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++) {
                inputf >> mat[convert_dimension_2D_1D(i, j)];
            }

        return 0;
    }

    string format_path(int i, int *pred) {
        string out("");
        int current_vertex = i;
        while (current_vertex != 0) {
            string s = std::to_string(current_vertex);
            std::reverse(s.begin(), s.end());
            out = out + s + ">-";
            current_vertex = pred[current_vertex];
        }
        out = out + std::to_string(0);
        std::reverse(out.begin(), out.end());
        return out;
    }

    int print_result(int *dist, int *pred) {
        std::ofstream outputf(outputfile, std::ofstream::out);
        outputf << dist[0];
        for (int i = 1; i < N; i++) {
            outputf << " " << dist[i];
        }
        for (int i = 0; i < N; i++) {
            outputf << "\n";
            if (dist[i] >= 1000000) {
                outputf << "NO PATH";
            } else {
                outputf << format_path(i, pred);
            }
        }
        outputf << endl;
        return 0;
    }
}//namespace utils


//------You may add helper functions and global variables here------

/*
 * function: find the local minimum for each block and store them to d_local_min and d_local_min_index
 * parameters: N: input size, *d_visit: array to record which vertex has been visited, *d_all_dist: array to store the distance,
 *        *d_local_min: array to store the local minimum value for each block, *d_local_min_index: array to store the local minimum index for each block
 */
__global__ void FindLocalMin(int N, int *d_visit, int *d_all_dist, int *d_local_min, int *d_local_min_index) {
    int num_vertices;
    if (N % (gridDim.x*blockDim.x) == 0){
        num_vertices = N/(gridDim.x * blockDim.x);
    } 
    else{
        num_vertices = N/(gridDim.x * blockDim.x) + 1;
    }

    __shared__ int t_min[1024];
    __shared__ int t_index[1024];

    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    t_min[threadIdx.x] = INT_MAX;
    t_index[threadIdx.x] = -1;

    for (int i = 0; i < num_vertices; i++){
        int gindex = threadID * num_vertices + i;
        if (gindex < N){
            if (!d_visit[gindex]){
                if (d_all_dist[gindex] < t_min[threadIdx.x]){
                    t_min[threadIdx.x] = d_all_dist[gindex];
                    t_index[threadIdx.x] = gindex;
                }
            }
        }
    }

    __syncthreads();

    if (threadIdx.x == 0){
        d_local_min[blockIdx.x] = INT_MAX;
        d_local_min_index[blockIdx.x] = -1;
        for (int i = 0; i < blockDim.x; i++){
            if(t_min[i] < d_local_min[blockIdx.x]){
                d_local_min[blockIdx.x] = t_min[i];
                d_local_min_index[blockIdx.x] = t_index[i];
            }
        }
    }

    __syncthreads();

}

/*
 * function: update the global minimum value(and index), store them to a global memory address
 * parameters: *global_min: memory address to store the global min value, *global_min_index: memory address to store the global min index
 *        *d_local_min: array stores the local min value od each block, *d_local_min_index: array stores the local min index of each block
 *        *d_visit: array stores the status(visited/un-visited) for each vertex
 */
__global__ void
UpdateGlobalMin(int *global_min, int *global_min_index, int *d_local_min, int *d_local_min_index, int *d_visit) {
    if (blockIdx.x == 0){
        *global_min = INT_MAX;
        *global_min_index = -1;
        for (int i = 0; i < gridDim.x; i++){
            if (d_local_min[i] < *global_min){
                *global_min = d_local_min[i];
                *global_min_index = d_local_min_index[i];
            }
        }
    }
}

/*
 * function: update the shortest path for every un-visited vertices
 * parameters: N: input size, *mat: input matrix, *d_visit: array stores the status(visited/un-visited) for each vertex
 *             *d_all_dist: array stores the shortest distance for each vertex, *d_all_pred: array stores the predecessors
 *             *global_min: memory address that stores the global min value, *global_min_index: memory address that stores the global min index
 */
__global__ void
UpdatePath(int N, int *mat, int *d_visit, int *d_all_dist, int *d_all_pred, int *global_min, int *global_min_index) {

    int num_vertices;
    if (N % (gridDim.x*blockDim.x) == 0){
        num_vertices = N/gridDim.x/blockDim.x;
    } 
    else{
        num_vertices = N/gridDim.x/blockDim.x + 1;
    }

    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    //if (blockIdx.x == 0 && threadIdx.x == 0)
    d_visit[*global_min_index] = 1;
    int u = *global_min_index;

    for (int i = 0; i < num_vertices; i++){
        int gindex = threadID * num_vertices + i;
        if(gindex < N){
            if (!d_visit[gindex]) {
                int new_dist = d_all_dist[u] + mat[u * N + gindex];
                if (new_dist < d_all_dist[gindex]) {
                    d_all_dist[gindex] = new_dist;
                    d_all_pred[gindex] = u;
                }
            }
        }
    } 
}

//Do not change anything below this line
void dijkstra(int N, int p, int *mat, int *all_dist, int *all_pred) {

    //threads number for each block should smaller than or equal to 1024
    assert(p <= 1024);

    //we restrict this value to 8, DO NOT change it!
    int blocksPerGrid = 8;

    //NOTICE: (p * 8) may LESS THAN N
    int threadsPerBlock = p;

    dim3 blocks(blocksPerGrid);
    dim3 threads(threadsPerBlock);


    //allocate memory
    int *h_visit;
    int *d_mat, *d_visit, *d_all_dist, *d_all_pred, *d_local_min, *d_local_min_index;
    int *d_global_min, *d_global_min_index;

    h_visit = (int *) calloc(N, sizeof(int));
    hipMalloc(&d_mat, sizeof(int) * N * N);
    hipMalloc(&d_visit, sizeof(int) * N);
    hipMalloc(&d_all_dist, sizeof(int) * N);
    hipMalloc(&d_all_pred, sizeof(int) * N);
    hipMalloc(&d_local_min, sizeof(int) * blocksPerGrid);
    hipMalloc(&d_local_min_index, sizeof(int) * blocksPerGrid);
    hipMalloc(&d_global_min, sizeof(int));
    hipMalloc(&d_global_min_index, sizeof(int));

    //initialization and copy data from host to device
    for (int i = 0; i < N; i++) {
        all_dist[i] = mat[i];
        all_pred[i] = 0;
        h_visit[i] = 0;
    }
    h_visit[0] = 1;

    hipMemcpy(d_mat, mat, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_all_dist, all_dist, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_all_pred, all_pred, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_visit, h_visit, sizeof(int) * N, hipMemcpyHostToDevice);

    //dijkstra iterations
    for (int iter = 1; iter < N; iter++) {
        FindLocalMin <<< blocks, threads >>> (N, d_visit, d_all_dist, d_local_min, d_local_min_index);
        //CHECK(hipDeviceSynchronize()); //only for debug
        UpdateGlobalMin <<< blocks, threads >>>
                                    (d_global_min, d_global_min_index, d_local_min, d_local_min_index, d_visit);
        //CHECK(hipDeviceSynchronize()); //only for debug
        UpdatePath << < blocks, threads >> >
                                (N, d_mat, d_visit, d_all_dist, d_all_pred, d_global_min, d_global_min_index);
        //CHECK(hipDeviceSynchronize()); //only for debug
    }

    //copy results from device to host
    hipMemcpy(all_dist, d_all_dist, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(all_pred, d_all_pred, sizeof(int) * N, hipMemcpyDeviceToHost);

    //free memory
    free(h_visit);
    hipFree(d_mat);
    hipFree(d_visit);
    hipFree(d_all_dist);
    hipFree(d_all_pred);
    hipFree(d_local_min);
    hipFree(d_local_min_index);
    hipFree(d_global_min);
    hipFree(d_global_min_index);

}

int main(int argc, char **argv) {
    assert(utils::parse_args(argc, argv) == 0);
    assert(utils::read_file(utils::filename) == 0);

    //`all_dist` stores the distances and `all_pred` stores the predecessors
    int *all_dist;
    int *all_pred;
    all_dist = (int *) calloc(utils::N, sizeof(int));
    all_pred = (int *) calloc(utils::N, sizeof(int));

    //time counter
    timeval start_wall_time_t, end_wall_time_t;
    float ms_wall;

    hipDeviceReset();

    //start timer
    gettimeofday(&start_wall_time_t, nullptr);
    dijkstra(utils::N, utils::num_threads, utils::mat, all_dist, all_pred);
    CHECK(hipDeviceSynchronize());

    //end timer
    gettimeofday(&end_wall_time_t, nullptr);
    ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000
               + end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

    std::cerr << "Time(ms): " << ms_wall << endl;

    utils::print_result(all_dist, all_pred);

    free(utils::mat);
    free(all_dist);
    free(all_pred);

    return 0;
}
